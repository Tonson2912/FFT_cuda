#include"func_header.h"
#include"fft_1D_cuda.cuh"

template<typename T>
class fft_2D_cuda :public func_header<T>
{
public:
	fft_2D_cuda(T *dataImage, unsigned int height,unsigned int width, unsigned int batch) : _dataImage(dataImage),_height(height), _width(width), _batch(batch) {
		cuda_init();
	}
	void get_dft_result(T* dft_result)
	{
		compute(dft_result);
	}
	void reset()
	{
		_dataImage = nullptr;
	}
	void del_object() {
		delete this;
	}

private:
	~fft_2D_cuda()
	{
		cuda_free();
		hipfftDestroy(_fftPlanFwd);
	}
	inline int iDivide(int a, int b) {
		return a % b != 0 ? a / b + 1 : a / b;
	}
	void cuda_init();
	void compute(T *dft_result);
	void cuda_free();
	unsigned int _height;
	unsigned int _width;
	unsigned int _batch;
	T* _dataImage;//�ⲿ�����������û���ָ�룬��������ָ�롣
	//gpu�豸�ڴ�
	T* _d_dataImage;
	hipfftDoubleComplex* _d_dataImage_comp;
	T* _d_dft_result;
	hipfftDoubleComplex* _d_dft_result_comp;
	hipfftHandle _fftPlanFwd;
};

template<typename T>
void fft_2D_cuda<T>::cuda_init()
{
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dataImage, sizeof(T)*_height*_width*_batch));
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dataImage_comp, sizeof(hipfftDoubleComplex)*_height*_width*_batch));
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dft_result, sizeof(T)*_height*_width*_batch));
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dft_result_comp, sizeof(hipfftDoubleComplex)*_height*_width*_batch));
	int n[2] = { _height,_width };
	int inembed[] = { _height,_width };
	int onembed[] = { _height,_width };
	CUFFT_CUDA_ERROR(hipfftPlanMany(&_fftPlanFwd, 2, n, inembed, 1, _height*_width, onembed, 1, _height*_width, HIPFFT_Z2Z, _batch));
}

template<typename T>
void fft_2D_cuda<T>::compute(T *dft_result)
{
	RUNTIME_CUDA_ERROR(hipMemcpy(_d_dataImage, _dataImage, sizeof(T)*_height*_width*_batch, hipMemcpyHostToDevice));

	//���и���Ҷ�任
	cufft_R2C_kernel << <iDivide(_height*_width*_batch, 512), 512 >> > (_d_dataImage, _d_dataImage_comp, _height*_width*_batch);

	CUFFT_CUDA_ERROR(hipfftExecZ2Z(_fftPlanFwd, _d_dataImage_comp, _d_dft_result_comp, HIPFFT_FORWARD));
	//READ_CUDA_DATA_COMP(_d_dft_result_comp, _length*_batch);

	cufft_comp_asb_kernel << <iDivide(_height*_width*_batch, 512), 512 >> > (_d_dft_result_comp, _d_dft_result, _height*_width*_batch);
	RUNTIME_CUDA_ERROR(hipGetLastError());


	RUNTIME_CUDA_ERROR(hipMemcpy(dft_result, _d_dft_result, sizeof(T)*_height*_width*_batch, hipMemcpyDeviceToHost));
}

template<typename T>
void fft_2D_cuda<T>::cuda_free()
{
	CUDA_FREE(_d_dataImage);
	CUDA_FREE(_d_dataImage_comp);
	CUDA_FREE(_d_dft_result_comp);
	CUDA_FREE(_d_dft_result);
	CUFFT_CUDA_ERROR(hipfftDestroy(_fftPlanFwd));
}

func_header<float> * get_fft_2D_GPU(float *dataImage, unsigned int height,unsigned int width, unsigned int batch)
{
	return new fft_2D_cuda<float>(dataImage, height, width, batch);
}