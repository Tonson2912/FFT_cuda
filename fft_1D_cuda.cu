#include"func_header.h"
#include"fft_1D_cuda.cuh"


template<typename T>
class fft_1D_cuda :public func_header<T>
{
public:
	fft_1D_cuda(T *dataline, unsigned int length,unsigned int batch) :_length(length), _dataline(dataline),_batch(batch) {
		cuda_init();
	}
	void get_dft_result(T* dft_result) 
	{
		compute(dft_result);
	}
	void reset()
	{
		_dataline = nullptr;
		_length = 0;
	}
	void del_object() {
		delete this;
	}

private:
	~fft_1D_cuda()
	{
		cuda_free();
		hipfftDestroy(_fftPlanFwd);
	}
	inline int iDivide(int a, int b) {
		return a % b != 0 ? a / b + 1 : a / b;
	}
	void cuda_init();
	void compute(T *dft_result);
	void cuda_free();
	unsigned int _length;
	unsigned int _batch;
	T* _dataline;//�ⲿ�����������û���ָ�룬��������ָ�롣
	//gpu�豸�ڴ�
	T* _d_dataline;
	hipfftDoubleComplex* _d_dataline_comp;
	T* _d_dft_result;
	hipfftDoubleComplex* _d_dft_result_comp;
	hipfftHandle _fftPlanFwd;
};

template<typename T>
void fft_1D_cuda<T>::cuda_init()
{
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dataline, sizeof(T)*_length*_batch));
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dataline_comp, sizeof(hipfftDoubleComplex)*_length*_batch));
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dft_result, sizeof(T)*_length*_batch));
	RUNTIME_CUDA_ERROR(hipMalloc(&_d_dft_result_comp, sizeof(hipfftDoubleComplex)*_length*_batch));
	int n[1] = { _length };
	int inembed[1] = { 0 };
	int onembed[1] = { 0 };
	CUFFT_CUDA_ERROR(hipfftPlanMany(&_fftPlanFwd, 1, n, inembed, 1, _length, onembed, 1, _length, HIPFFT_Z2Z, _batch));
}

template<typename T>
void fft_1D_cuda<T>::compute(T *dft_result)
{
	RUNTIME_CUDA_ERROR(hipMemcpy(_d_dataline, _dataline, sizeof(T)*_length*_batch, hipMemcpyHostToDevice));

	//���и���Ҷ�任
	cufft_R2C_kernel << <iDivide(_length*_batch, 256), 256 >> > (_d_dataline, _d_dataline_comp, _length*_batch);

	CUFFT_CUDA_ERROR(hipfftExecZ2Z(_fftPlanFwd, _d_dataline_comp, _d_dft_result_comp, HIPFFT_FORWARD));
	//READ_CUDA_DATA_COMP(_d_dft_result_comp, _length*_batch);

	cufft_comp_asb_kernel << <iDivide(_length*_batch, 256), 256 >> > (_d_dft_result_comp, _d_dft_result, _length*_batch);
	RUNTIME_CUDA_ERROR(hipGetLastError());


	RUNTIME_CUDA_ERROR(hipMemcpy(dft_result, _d_dft_result, sizeof(T)*_length*_batch, hipMemcpyDeviceToHost));
}

template<typename T>
void fft_1D_cuda<T>::cuda_free()
{
	CUDA_FREE(_d_dataline);
	CUDA_FREE(_d_dft_result);
	CUDA_FREE(_d_dataline_comp);
	CUDA_FREE(_d_dft_result_comp);
	CUFFT_CUDA_ERROR(hipfftDestroy(_fftPlanFwd));
}

func_header<float> * get_fft_1D_GPU(float *dataline, unsigned int length,unsigned int batch)
{
	return new fft_1D_cuda<float>(dataline, length, batch);
}